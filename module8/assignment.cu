#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <hipfft/hipfft.h>

typedef float2 Complex;


const uint16_t imgSize = 128;

/*
 * Kernel to calculate relative position from center for a given pixel in a vectorized 2D image/array
 */
__global__ void calcRelativePosition(float *pos, int arraySize)
{
	uint idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint col = idx%arraySize;
    uint row = idx/arraySize;

	//int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x)+ threadIdx.x;

	int pixelPos = (row*arraySize)+col;

	uint rel_row = row-arraySize/2;
	uint rel_col = col-arraySize/2;

	float relative = sqrt((float)((rel_col*rel_col)+ (rel_row*rel_row)));
	pos[pixelPos] = relative;

	//printf("xPos: %3u yPos: %3u \nPixel Distance from Center: %3.3f\n", col, row, relative);

}

/*
 * Kernel to multiply two complex matrices
 */
__global__ void ComplexMUL(Complex *a, Complex *b)
{
	// Adapted from cufft_example.cu file.
	int i = threadIdx.x + blockIdx.x*blockDim.x;
    a[i].x = a[i].x * b[i].x - a[i].y*b[i].y;
    a[i].y = a[i].x * b[i].y + a[i].y*b[i].x;
}


__host__ void loadImage() {
	// Just loading array of random values.

}


__host__ void driver() {
	const int totalPixels = imgSize*imgSize;
	const int blockSize = 256;
	const int numBlocks = totalPixels/blockSize;

	const int size_bytes = totalPixels*sizeof(float);


	float *h_relPos;
	float *d_relPos;

	h_relPos    = (float*)malloc(size_bytes);
	hipMalloc((void**)&d_relPos, size_bytes);

	calcRelativePosition<<<numBlocks,blockSize>>>(d_relPos, imgSize);

	hipMemcpy(h_relPos, d_relPos, size_bytes, hipMemcpyDeviceToHost);

    int center = (imgSize/2);
    
    printf("Relative Position: %3.3f for (%d,%d)", h_relPos[center*imgSize+(center+20)], center+20, center);


	hipFree(d_relPos);
}



int main(int argc, char** argv)
{
    driver();
	return 0;
}
