#include "hip/hip_runtime.h"
/*
 * filterGen.cu
 *
 *  Created on: Apr 22, 2019
 *      Author: zetser
 */

#include "filterGen.h"

__constant__ float  k;
__constant__ float  pix_DX;
__constant__ float  pix_DY;
__constant__ int    pixWidth;
__constant__ int    pixHeight;
__constant__ float  const_zDist;

__global__ void genFilter(complex *filt) {

    // ===============================================================
    // This part will need to change if I change block/thread paradigm
    uint idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint row = idx/pixWidth;
    uint col = idx%pixWidth;
    // ===============================================================

    float x = ((float)col - pixWidth/2.0)  * pix_DX;
	float y = ((float)row - pixHeight/2.0) * pix_DY;

	// Want to store this in global memory and use it for the filter in another kernel.
    filt[idx].x =  ( k/(2*(M_PI)*const_zDist) ) * sin( (k/(2*const_zDist)) * ((x*x)+(y*y)+(const_zDist*const_zDist))  );
    filt[idx].y = -( k/(2*(M_PI)*const_zDist) ) * cos( (k/(2*const_zDist)) * ((x*x)+(y*y)+(const_zDist*const_zDist))  );

//    if(row==256 && col==256) {
//        printf("(%d,%d):\t%3.5f + j%3.5f\n", row, col, filt[idx].x, filt[idx].y);
//    }
}

__global__ void ComplexMUL(complex *a, complex *b)
{
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    float cx = a[i].x * b[i].x - a[i].y * b[i].y;
    float cy = a[i].x * b[i].y + a[i].y * b[i].x;

    a[i].x = cx;
    a[i].y = cy;

//    printf("%f + j%f\n", a[i].x, a[i].y);
}

__global__ void fftshift(complex *in_img, complex *out_img) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    uint oldRow = idx/pixWidth;
    uint oldCol = idx%pixWidth;

    uint newRow = 0;
    uint newCol = 0;

    if(oldRow<pixHeight && oldCol<pixWidth) {
        if( oldRow<(pixHeight/2-1) ) {
            newRow = oldRow + (pixHeight/2) - 1;
        }
        else {
            newRow = oldRow - (pixHeight/2) - 1;
        }

        if( oldCol<(pixWidth/2-1) ) {
            newCol = oldCol + (pixWidth/2) - 1;
        }
        else {
            newCol = oldCol - (pixWidth/2) - 1;
        }

        out_img[newRow*pixWidth+newCol] = in_img[oldRow*pixWidth+oldCol];


    }

}



__global__ void createHolo(complex* fzp, float* out) {
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;

    out[i] = ( (1+fzp[i].x)*(1+fzp[i].x) ) - ( fzp[i].y * fzp[i].y );
//    fzp[i].x += 1;
//    out[i] = hipCabsf(fzp[i]);
//    printf("%f + j%f\n", fzp[i].x,fzp[i].y);
}

void calcSpatImpulseResponse(cimg_library::CImg<unsigned char> img, float distance) {

//    cimg_library::CImgDisplay main_disp(img,"Original");
//
//    while (!main_disp.is_closed()) {
//        main_disp.wait();
//    }

    const int width = img.width();
    const int height = img.height();
    const int numPix = width*height;
    const float waveLength = 550*pow(10,-6);        // 550nm -> green light
    const float kVal = 2*(M_PI)/waveLength;
    const float imWidth  = 0.001;                   // 1mm
    const float imHeight = 0.001;                   // 1mm
    const float dx = imWidth/(width-1);
    const float dy = imHeight/(height-1);


// ============================================================================
    // Let h_zDist be the distance from the object plane to the image plane.
    // Constant for simplicity for now.
    float *h_zDist;

    const int zSize = sizeof(float);
    h_zDist = (float*)malloc(zSize);
    h_zDist[0] = distance;
    /* For when each pixel may have a different distance
    const int zSize = width*height*sizeof(float);
    h_zDist = new float[width*height];
    for(int i=0; i<numPix; i++)
        h_zDist[i] = .001;

    float *d_zDist;     // Z distance specified for each pixel, currently setting as 1 value, but should be variable for future changes.
    hipMalloc((void**)&d_zDist, zSize);
    hipMemcpy(d_zDist, h_zDist, zSize, hipMemcpyHostToDevice);
    */
// ============================================================================
// Allocate device memory
    const int compImgSize = numPix*sizeof(hipfftComplex);

    // Allocate and copy over input Image

    complex *h_image = new complex[numPix];
    for(int i=0; i<height; i++) {
        for(int j=0; j<width; j++) {
            h_image[(i*width+j)].x = (img(j, i, 0)/255.0);     // Only performing filter on one color channel
            h_image[(i*width+j)].y = 0;
//            printf("Image value: %f\n", h_image[(i*width+j)].x);
        }
    }
    hipfftComplex *d_image;
    hipMalloc((void**)&d_image, compImgSize);
    hipMemcpy(d_image, h_image, compImgSize, hipMemcpyHostToDevice);

    std::cout << "Did I make it?" << std::endl;
    // We will be generating the filter based on position and will never pull it out of GPU memory

    hipfftComplex *d_filter;   // filter that will be calculated based on position by GPU
    hipMalloc((void**)&d_filter, compImgSize);
// ============================================================================
// Copy values to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(k),           &kVal,    sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pix_DX),      &dx,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pix_DY),      &dy,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pixHeight),   &height,  sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(pixWidth),    &width,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(const_zDist), h_zDist,  sizeof(float));
// ============================================================================
// Generate impulse response of free space propagation
    const int blockSize = 1024;
    const int numBlocks = numPix/blockSize;

    std::cout << "Creating filter" << std::endl;
    genFilter <<<numBlocks, blockSize>>>(d_filter);     // Leave filter in GPU global memory for use in cuFFT

// ============================================================================
// Perform FFT on filter and input image
    hipfftHandle plan;
    hipfftPlan2d(&plan, width, height, HIPFFT_C2C);

    // Perform FFTs
    hipfftExecC2C(plan, (hipfftComplex *)d_filter,     (hipfftComplex *)d_filter,     HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)d_image,      (hipfftComplex *)d_image,      HIPFFT_FORWARD);

    ComplexMUL <<<numBlocks, blockSize>>>(d_image, d_filter);

    hipfftExecC2C(plan, (hipfftComplex *)d_image,      (hipfftComplex *)d_image,      HIPFFT_BACKWARD);
//    fftshift_1D <<<numBlocks, blockSize>>>(d_image);

    // Free filter memory, as we no longer need it.
    hipFree(d_filter);
// ============================================================================
// Convert Fresnel propagation into hologram
    float* h_holo;
    h_holo = new float[numPix];

    float* d_holo;
    hipMalloc((void**)&d_holo, sizeof(float)*numPix);

    createHolo <<<numBlocks, blockSize>>>(d_image, d_holo);


    hipMemcpy(h_holo, d_holo, sizeof(float)*numPix, hipMemcpyDeviceToHost);

//    float* myMax = std::max_element(h_holo, h_holo+(numPix-1));
    cimg_library::CImg<float> holoImg(width, height, 1, 1, 0);
    for(int i=0; i<height; i++) {
        for(int j=0; j<width; j++) {
            holoImg(j, i, 0) = h_holo[i*width+j];
        }
    }


    cimg_library::CImgDisplay finalDisp(holoImg,"Hologram Image");
    while (!finalDisp.is_closed()) {
        finalDisp.wait();
    }
//    img.save_bmp("origLena.bmp");
    holoImg.save_bmp("createdHologram.png");

    hipFree(d_image);
    hipFree(d_holo);
}